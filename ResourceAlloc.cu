#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define max_N 100000
#define max_P 30
#define BLOCKSIZE 1024
#define SLOTS 24

using namespace std;

//*******************************************
// Write down the kernels here
__global__ void find_success(int N, int *d_req_id, int *d_req_fac, int *d_req_cen, int *d_req_start, int *d_req_slots, int *d_capacity, int *d_center_start, int *d_facility, int *d_tot_req, int *d_succ_req){

  int id = blockIdx.x * blockDim.x + threadIdx.x; // Finding id

  if(id<N){
    int cnt_capacity[max_P*SLOTS]; // array that checks maximum capacity for the facility
    int k = 0;
    // Initializing the cnt_capacity array with zero
    while(k < d_facility[id]*SLOTS){
      cnt_capacity[k] = 0;
      k++;
    }

    // Finding request is successfull or not
    int i = d_tot_req[id];
    while(i<d_tot_req[id+1]){
      int s_index = d_req_start[i] - 1 + d_req_fac[i]*SLOTS;
      int e_index = d_req_slots[i] + s_index;
      int l = s_index;
      int cnt = 0;

      while(l < e_index){
        if(cnt_capacity[l] < d_capacity[d_center_start[id]+d_req_fac[i]] && e_index <= d_req_fac[i]*SLOTS + SLOTS){
          cnt++;
        }
        l++;
      }

      if(cnt == e_index - s_index){
        l = s_index;
        while(l < e_index){
          cnt_capacity[l]++;
          l++;
        }
        d_succ_req[id]++;
      }
      i++;
    }
  }
}

//***********************************************

// Function to sort on center ids
int partition_array(int *req_id, int *req_cen, int *req_fac, int *req_start, int *req_slots, int S, int R){

  int i = (S-1);
  for (int j=S; j<=R-1; j++)  
  {  
    if ((req_cen[j] < req_cen[R]) || ((req_cen[j] == req_cen[R]) && req_id[j] < req_id[R])){  
      i++; 
      swap(req_cen[j], req_cen[i]);
      swap(req_fac[j], req_fac[i]);
      swap(req_id[j], req_id[i]);
      swap(req_slots[j], req_slots[i]);
      swap(req_start[j], req_start[i]); 
    }  
  }  
  swap(req_cen[R], req_cen[i+1]);
  swap(req_fac[R], req_fac[i+1]);
  swap(req_id[R], req_id[i+1]);
  swap(req_slots[R], req_slots[i+1]);
  swap(req_start[R], req_start[i+1]); 
  return (i+1); 
} 

// Sorting request array wrt req_cen
void sort_center(int *req_id, int *req_cen, int *req_fac, int *req_start, int *req_slots, int S, int R){
  if (S<R) {  
    int p = partition_array(req_id, req_cen, req_fac, req_start, req_slots, S, R);
    sort_center(req_id, req_cen, req_fac, req_start, req_slots, S, p - 1);  
    sort_center(req_id, req_cen, req_fac, req_start, req_slots, p + 1, R);  
  }  
}

int main(int argc,char **argv)
{
	// variable declarations...
    int N,*centre,*facility,*capacity,*fac_ids, *succ_reqs, *tot_reqs;

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &N ); // N is number of centres
	
    // Allocate memory on cpu
    centre=(int*)malloc(N * sizeof (int));  // Computer  centre numbers
    facility=(int*)malloc(N * sizeof (int));  // Number of facilities in each computer centre
    fac_ids=(int*)malloc(max_P * N  * sizeof (int));  // Facility room numbers of each computer centre
    capacity=(int*)malloc(max_P * N * sizeof (int));  // stores capacities of each facility for every computer centre 

    int success = 0;  // total successful requests
    int fail = 0;   // total failed requests
    tot_reqs = (int *)malloc(N*sizeof(int));   // total requests for each centre
    succ_reqs = (int *)malloc(N*sizeof(int)); // total successful requests for each centre

    // Input the computer centres data
    int k1=0 , k2 = 0;
    for(int i=0;i<N;i++)
    {
      fscanf( inputfilepointer, "%d", &centre[i] );
      fscanf( inputfilepointer, "%d", &facility[i] );
      
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &fac_ids[k1] );
        k1++;
      }
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &capacity[k2]);
        k2++;     
      }
    }

    // variable declarations
    int *req_id, *req_cen, *req_fac, *req_start, *req_slots;   // Number of slots requested for every request
    
    // Allocate memory on CPU 
	  int R;
	  fscanf( inputfilepointer, "%d", &R); // Total requests
    req_id = (int *) malloc ( (R) * sizeof (int) );  // Request ids
    req_cen = (int *) malloc ( (R) * sizeof (int) );  // Requested computer centre
    req_fac = (int *) malloc ( (R) * sizeof (int) );  // Requested facility
    req_start = (int *) malloc ( (R) * sizeof (int) );  // Start slot of every request
    req_slots = (int *) malloc ( (R) * sizeof (int) );   // Number of slots requested for every request
    
    // Input the user request data
    for(int j = 0; j < R; j++)
    {
       fscanf( inputfilepointer, "%d", &req_id[j]);
       fscanf( inputfilepointer, "%d", &req_cen[j]);
       fscanf( inputfilepointer, "%d", &req_fac[j]);
       fscanf( inputfilepointer, "%d", &req_start[j]);
       fscanf( inputfilepointer, "%d", &req_slots[j]);
       tot_reqs[req_cen[j]]+=1;  
    }
		
    // call for sorting the array wrt center id
    sort_center(req_id, req_cen, req_fac, req_start, req_slots, 0, R-1);
    
    // variable declarations on CPU
    int *center_start;
    int *total_req;
    // variable declarations on GPU
    int *d_req_id, *d_req_fac, *d_req_cen, *d_req_start, *d_req_slots, *d_capacity, *d_center_start, *d_facility, *d_tot_req, *d_succ_req;

    // Allocate memory on CPU
    center_start = (int *)malloc(N*sizeof(int));
    total_req = (int *)malloc((N+1)*sizeof(int));

    //Finding starting index of each center and starting index of request of each center
    int sum = 0;
    total_req[0] = 0;
    for(int i=0; i<N; i++){
      total_req[i+1] = total_req[i] + tot_reqs[i];
      center_start[i] = sum;
      sum += facility[i];
    }

    // Allocate memory on GPU
    hipMalloc(&d_req_id, R*sizeof(int));
    hipMalloc(&d_req_fac, R*sizeof(int));
    hipMalloc(&d_req_cen, R*sizeof(int));
    hipMalloc(&d_req_start, R*sizeof(int));
    hipMalloc(&d_req_slots, R*sizeof(int));
    hipMalloc(&d_capacity, max_P*N*sizeof(int));
    hipMalloc(&d_center_start, N*sizeof(int));
    hipMalloc(&d_facility, N*sizeof(int));
    hipMalloc(&d_tot_req, (N+1)*sizeof(int));
    hipMalloc(&d_succ_req, N*sizeof(int));

    // copying memory from CPU to GPU
    hipMemcpy(d_req_id, req_id, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_fac, req_fac, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_cen, req_cen, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_start, req_start, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_slots, req_slots, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_capacity, capacity, max_P*N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_center_start, center_start, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_facility, facility, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_tot_req, total_req, (N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_succ_req, succ_reqs, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_succ_req, 0, N*sizeof(int));


    //*********************************
    // Call the kernels here
    int blocksize = ceil((float)N/1024);
    find_success<<<blocksize, BLOCKSIZE>>>(N, d_req_id, d_req_fac, d_req_cen, d_req_start, d_req_slots, d_capacity, d_center_start, d_facility, d_tot_req, d_succ_req);
    hipMemcpy(succ_reqs, d_succ_req, N*sizeof(int), hipMemcpyDeviceToHost);
    //********************************

    // Calculating total number of success and failure
    for(int i=0; i<N; i++){
      success += succ_reqs[i];
    }
    fail = total_req[N] - success;

    // Output
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    fprintf( outputfilepointer, "%d %d\n", success, fail);
    for(int j = 0; j < N; j++)
    {
        fprintf( outputfilepointer, "%d %d\n", succ_reqs[j], tot_reqs[j]-succ_reqs[j]);
    }
    fclose( inputfilepointer );
    fclose( outputfilepointer );
    hipDeviceSynchronize();
	return 0;
}